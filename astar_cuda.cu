#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "astar_cuda.h"

// CUDA Kernel to calculate Euclidean distance
__global__ void calculateDistancesKernel(float *d_x, float *d_y, float goal_x, float goal_y, float *d_distances, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float dx = d_x[idx] - goal_x;
        float dy = d_y[idx] - goal_y;
        d_distances[idx] = sqrtf(dx * dx + dy * dy);
    }
}

extern "C" void calculateDistances(float *h_x, float *h_y, float goal_x, float goal_y, float *h_distances, int n) {
    float *d_x, *d_y, *d_distances;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_distances, n * sizeof(float));

    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    calculateDistancesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, goal_x, goal_y, d_distances, n);

    hipMemcpy(h_distances, d_distances, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_distances);
}